#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


#define N 10000000
#define MAX_ERR 1e-6


__global__ void vector_add(float *out, float *a, float *b, int n)
{
    for (int i = 0; i < n; i++)
    {
        out[i] = a[i] + b[i];
    }
}

int main()
{
    printf("Initializing host and device variables...\n");
    float *a, *b, *out;
    float *da, *db, *dout;

    printf("Allocating host memory...\n");
    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    printf("Initializing host array...\n");
    for (int i = 0; i < N; i++)
    {
        a[i] = 1.0f; b[i] = 2.0f;
    }

    printf("Allocating device memory...\n");
    hipMalloc((void**)&da, sizeof(float) * N);
    hipMalloc((void**)&db, sizeof(float) * N);
    hipMalloc((void**)&dout, sizeof(float) * N);    

    printf("Transferring data to device memory...\n");
    hipMemcpy(da, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(float) * N, hipMemcpyHostToDevice);

    printf("Adding vectors...\n");
    vector_add<<<1,1>>>(dout, da, db, N);

    printf("Transferring data back to host memory...\n");
    hipMemcpy(out, dout, sizeof(float) * N, hipMemcpyDeviceToHost);

    // printf("Verifying output...\n");
    // for (int i = 0; i < N; i++)
    // {
    //     assert(fabs(out[i] - a[i] -b[i]) < MAX_ERR);
    // }

    printf("out[0] = %f\n", out[0]);

    printf("Deallocating device memory...\n");
    hipFree(da);
    hipFree(db);
    hipFree(dout);

    printf("Deallocating host memory...\n");
    free(a);
    free(b);
    free(out);
}