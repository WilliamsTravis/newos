#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <sys/wait.h>

#define N 2000000000
#define MAX_ERR 1e-8


__global__ void vector_add(float *out, float *a, float *b, long int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n)
    {
    
        // Calculate pi
        int r[2800 + 1];
        int i;
        long int k;
        int e, d;
        for (i = 0; i < 2800; i++) {
            r[i] = 2000;
        }
        for (k = 9000000000000000000; k > 0; k -= 14) {
            d = 0;
            i = k;
            for (;;) {
                d += r[i] * 10000;
                e = 2 * i - 1;
    
                r[i] = d % e;
                d /= e;
                i--;
                if (i == 0) break;
                d *= i;
            }
            float x = d % 10000;
        }

        // Calculate a big factorial
        long int number = 9000000000000000000;
        int fact = 1;    
        for(long int y = 1; y <= number; y++)
        {    
            fact = fact * y;
            long int woo = fact + fact;
            woo *= 2;
            woo *= 2;
            woo *= 2;
            woo *= 2;
            woo *= 2;
            woo *= woo;
            
            woo += woo;            
        }    

        // Do whatever this is
        float ta = a[tid];
        float tb = b[tid];
        ta *= tb + ta;
        
        ta = ta * 2 + tb * 5;
        tb = tb * 5 + tb * 2;
        out[tid] = ta + tb;
    }
}

int main()
{
    printf("Initializing host and device variables...\n");
    float *a, *b, *out;
    float *da, *db, *dout;

    printf("Allocating host memory...\n");
    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    printf("Initializing host array...\n");

    #pragma omp parallel for num_threads(N)
    for (int i = 0; i < N; i++)
    {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    printf("Allocating device memory...\n");
    hipMalloc((void**)&da, sizeof(float) * N);
    hipMalloc((void**)&db, sizeof(float) * N);
    hipMalloc((void**)&dout, sizeof(float) * N);    

    printf("Transferring data to device memory...\n");
    hipMemcpy(da, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(float) * N, hipMemcpyHostToDevice);

    printf("Adding vectors...\n");
    int block_size = 16;
    int grid_size = ((N + block_size) / block_size);
    vector_add<<<grid_size, block_size>>>(dout, da, db, N);

    printf("Transferring data back to host memory...\n");
    hipMemcpy(out, dout, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Verify output
//    for (int i = 0; i < N; i++)
  //  {
    //    assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    //}
    //printf("PASSED\n");

    printf("Deallocating device memory...\n");
    hipFree(da);
    hipFree(db);
    hipFree(dout);

    printf("Deallocating host memory...\n");
    free(a);
    free(b);
    free(out);

}
